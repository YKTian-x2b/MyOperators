
#include <hip/hip_runtime.h>
template <typename T>
__device__ __host__ std::enable_if_t<sizeof(T) == 1, T> constexpr next_power_of_two(T v)
{
    --v;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    return ++v;
}

template <typename T>
__device__ __host__ std::enable_if_t<sizeof(T) == 2, T> constexpr next_power_of_two(T v)
{
    --v;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    return ++v;
}

template <typename T>
__device__ __host__ std::enable_if_t<sizeof(T) == 4, T> constexpr next_power_of_two(T v)
{
    --v;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    return ++v;
}

template <typename T>
__device__ __host__ std::enable_if_t<sizeof(T) == 8, T> constexpr next_power_of_two(T v)
{
    --v;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v |= v >> 32;
    return ++v;
}