#include "hip/hip_runtime.h"
/*
template <typename TypeA, typename TypeB>
struct MixedGemmArchTraits<TypeA, TypeB, cutlass::arch::Sm80,
    typename cutlass::platform::enable_if<cutlass::platform::is_same<TypeA, cutlass::half_t>::value
        || cutlass::platform::is_same<TypeA, cutlass::bfloat16_t>::value>::type>
{
private:
    using LayoutDetails = LayoutDetailsB<TypeA, TypeB, cutlass::arch::Sm80>;

public:
    static constexpr int ThreadblockK = LayoutDetails::ThreadblockK;

    using OperatorClass = cutlass::arch::OpClassTensorOp;
    using AccType = float;
    using LayoutB = typename LayoutDetails::Layout;

    static constexpr int ElementsPerAccessA = 128 / cutlass::sizeof_bits<TypeA>::value;
    static constexpr int ElementsPerAccessB = LayoutDetails::ElementsPerAccess;
    static constexpr int ElementsPerAccessC = 128 / cutlass::sizeof_bits<TypeA>::value;
    using InstructionShape = cutlass::gemm::GemmShape<16, 8, 16>;

    using Operator = typename LayoutDetails::Operator;
};
*/

template <typename ActivationType, typename WeightType, typename ScaleZeroType, typename BiasType, typename OutputType,
    typename arch, cutlass::WeightOnlyQuantOp QuantOp, typename EpilogueTag, typename ThreadblockShape,
    typename WarpShape, int Stages>
void generic_mixed_gemm_kernelLauncher(ActivationType const* A, WeightType const* B, ScaleZeroType const* weight_scales,
    ScaleZeroType const* weight_zero_points, BiasType const* biases, float const alpha, OutputType* C, int m, int n,
    int k, int const group_size, tkc::CutlassGemmConfig gemm_config, char* workspace, size_t workspace_bytes,
    hipStream_t stream, int* occupancy = nullptr)
{ 
    using CutlassActivationType = cutlass::float_e4m3_t;    // __hip_fp8_e4m3_fnuz
    using CutlassWeightType = cutlass::uint4b_t;
    using CutlassScaleZeroType = half;
    using CutlassBiasType = half;
    using CutlassOutputType = half;
    // cutlass::WeightOnlyQuantOp::FINEGRAINED_SCALE_AND_ZEROS

    // We need separate config for each architecture since we will target different tensorcore instructions. For float,
    // we do not target TCs.
    using MixedGemmArchTraits
        = cutlass::gemm::kernel::MixedGemmArchTraits<CutlassActivationType, CutlassWeightType, arch>;
    using ElementAccumulator = typename MixedGemmArchTraits::AccType;

    constexpr int ElementsPerAccessC = 128 / cutlass::sizeof_bits<CutlassOutputType>::value;
    using EpilogueOp =
        typename tkc::Epilogue<CutlassOutputType, ElementsPerAccessC, ElementAccumulator, EpilogueTag>::Op;

    using Operator = typename MixedGemmArchTraits::Operator;
    using TaggedOperator = typename cutlass::arch::TagOperator<Operator, QuantOp>::TaggedOperator;

    using GemmKernel_ = typename cutlass::gemm::kernel::DefaultGemm<
    CutlassActivationType, cutlass::layout::RowMajor, MixedGemmArchTraits::ElementsPerAccessA, 
    CutlassWeightType, typename MixedGemmArchTraits::LayoutB, MixedGemmArchTraits::ElementsPerAccessB, 
    CutlassOutputType, cutlass::layout::RowMajor, ElementAccumulator,
        cutlass::arch::OpClassTensorOp, arch, ThreadblockShape, WarpShape,
        typename MixedGemmArchTraits::InstructionShape, EpilogueOp,
        typename cutlass::gemm::threadblock::GemmIdentityThreadblockSwizzle<>, Stages, true,
        TaggedOperator>::GemmKernel;

    using GemmKernel = cutlass::gemm::kernel::GemmFpAIntB<typename GemmKernel_::Mma, typename GemmKernel_::Epilogue,
        typename GemmKernel_::ThreadblockSwizzle,
        arch, // Ensure top level arch is used for dispatch
        GemmKernel_::kSplitKSerial>;

    if (occupancy != nullptr)
    {
        *occupancy = tensorrt_llm::cutlass_extensions::compute_occupancy_for_kernel<GemmKernel>();
        return;
    }

    using Gemm = cutlass::gemm::device::GemmUniversalBaseCompat<GemmKernel>;

    int const ldb = cutlass::platform::is_same<cutlass::layout::RowMajor, typename MixedGemmArchTraits::LayoutB>::value
        ? n
        : k * GemmKernel::kInterleave;

    if (weight_scales == nullptr)
    {
        throw std::runtime_error("Weight scales must always be set to a non-null value.");
    }
    if (group_size != 128)
    {
        throw std::runtime_error("Only group size 128 supported for fine grained W4A(fp)8 kernels.");
    }
    if (weight_zero_points == nullptr)
    {
        throw std::runtime_error("Weight zero pointer must be valid for scale and bias fine grained");
    }
   
    int const ld_scale_zero = cutlass::isFinegrained(QuantOp) ? n : 0;
    ElementAccumulator output_op_beta = (biases == nullptr) ? ElementAccumulator(0.f) : ElementAccumulator(1.f);
    typename Gemm::Arguments args({m, n, k}, group_size,
        {reinterpret_cast<CutlassActivationType*>(const_cast<ActivationType*>(A)), k},
        {reinterpret_cast<CutlassWeightType*>(const_cast<WeightType*>(B)), ldb},
        {reinterpret_cast<CutlassScaleZeroType*>(const_cast<ScaleZeroType*>(weight_scales)), ld_scale_zero},
        {reinterpret_cast<CutlassScaleZeroType*>(const_cast<ScaleZeroType*>(weight_zero_points)), ld_scale_zero},
        {reinterpret_cast<CutlassBiasType*>(const_cast<BiasType*>(biases)), 0},
        {reinterpret_cast<CutlassOutputType*>(C), n}, gemm_config.split_k_factor,
        {ElementAccumulator(alpha), output_op_beta});

    Gemm gemm;
    auto can_implement = gemm.can_implement(args);
    auto init_status = gemm.initialize(args, workspace, stream);
    auto run_status = gemm.run(stream);
}