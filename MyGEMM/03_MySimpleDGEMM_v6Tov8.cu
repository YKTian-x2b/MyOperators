#include "hip/hip_runtime.h"
#include "../Common.cuh"
#include "../Utils/atomicAdd.cu"
#include <iostream>
#include <random>
#include <ctime>
#include "hipblas.h"
#include <string>

// transpose操作 要求M/N/K都至少是16
// #define M 512
// #define N 512
// #define K 2048
#define M 256
#define N 256
#define K 512
// M/N/K要大于等于SMEM_PART
#define SMEM_PART 32

#define T_BLOCK_X 16
#define T_BLOCK_Y 16

const double alpha = 1.0;
const double beta = 0.0;

double A[M][K], B[K][N];
double C[M][N];

__host__ void assignData(){
    std::default_random_engine e;
    e.seed(time(0));
    std::uniform_real_distribution<double> u(1.5, 4.5);
    // std::cout << "Matrix A: " << std::endl;
    for(int i = 0; i < M; i++){
        for(int j = 0; j < K; j++){
            // A[i][j] = u(e);
            // A[i][j] = (i*2+j)*0.001; 
            A[i][j] = 1;   
            // std::cout << A[i][j] << " ";
        }
        // std::cout << std::endl;
    }
    // std::cout << std::endl; 
    // std::cout << "Matrix B: " << std::endl;
    for(int i = 0; i < K; i++){
        for(int j = 0; j < N; j++){
            // B[i][j] = u(e);
            // B[i][j] = (i*2+j)*0.001;   
            B[i][j] = 1; 
            // std::cout << B[i][j] << " ";
        }
        // std::cout << std::endl;
    }
    // std::cout << std::endl;  
    memset(C, 0, sizeof(double) * M * N);
}

__global__ void transpose(double * B, double * B_T){
    __shared__ double smem[T_BLOCK_Y][T_BLOCK_X];
    // 共享内存和线程块的结构是一样的，全局内存和网格的结构使用一样的。

    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    int gdx = gridDim.x, gdy = gridDim.y;

    int thread_idx_x = bix * bdx + tix;
    int thread_idx_y = biy * bdy + tiy;
    int thread_x_len = gdx * bdx;
    int thread_y_len = gdy * bdy;
    // 既是线程索引也是全局内存B的索引
    int thread_idx = thread_idx_y * thread_x_len + thread_idx_x;
    // B_T索引
    int tran_thread_idx = thread_idx_x * thread_y_len + thread_idx_y;

    // 行读列写
    smem[tiy][tix] = B[thread_idx];
    B_T[tran_thread_idx] = smem[tiy][tix];

}

__host__ void checkRes(double * B_T, double * B_T_blas, const int B_size){
    bool wrong = false;
    for(int i = 0; i < B_size; i++){
        double err = B_T[i] - B_T_blas[i];
        if(fabs(err) >=  0.01){
            wrong = true;
            std::cout << "err: " << B_T[i] << " - " << B_T_blas[i] << " = " << fabs(err) << std::endl;
        }
    }
    if(wrong == false){
        std::cout << "transpose success!" << std::endl;
    }
    else{
        std::cout << "transpose error!" << std::endl;
    }
}

__host__ void callTranspose(double * B, double * B_T, const int B_size){
    size_t B_Bytes = sizeof(double) * B_size;

    dim3 blockSize(T_BLOCK_X, T_BLOCK_Y);
    dim3 gridSize(N/T_BLOCK_X, K/T_BLOCK_Y);
    transpose<<<gridSize, blockSize>>>(B, B_T);
    hipDeviceSynchronize();
    double * h_B_T = (double*)malloc(B_Bytes);
    CHECK(hipMemcpy(h_B_T, B_T, B_Bytes, hipMemcpyDeviceToHost));
    // std::cout << "h_B_T: " << std::endl;
    // printMatrix(h_B_T, N, K);
    
    double * B_T_blas;
    CHECK(hipMalloc(&B_T_blas, B_Bytes));
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t stat = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, K, N, &alpha, B, N, &beta, nullptr, N, B_T_blas, K);
    hipDeviceSynchronize();
    double * h_B_T_blas = (double*)malloc(B_Bytes);
    CHECK(hipMemcpy(h_B_T_blas, B_T_blas, B_Bytes, hipMemcpyDeviceToHost));
    // std::cout << "h_B_T_blas: " << std::endl;
    // printMatrix(h_B_T_blas, N,  K);
 
    checkRes(h_B_T, h_B_T_blas, B_size);

    free(h_B_T);
    CHECK(hipFree(B_T_blas));
    hipblasDestroy(handle);
    free(h_B_T_blas);
}

__global__ void DGEMM_v6(double * A, double * B, double * C){
    // __shared__ double A_part[SMEM_PART][SMEM_PART], B_part[SMEM_PART][SMEM_PART];
    // __shared__ double C_part[SMEM_PART][N];
    // // 线程组索引
    // int tix = threadIdx.x, tiy = threadIdx.y;
    // int bix = blockIdx.x, biy = blockIdx.y;
    // int bdx = blockDim.x, bdy = blockDim.y;
    // int gdx = gridDim.x, gdy = gridDim.y;

    // int thread_idx_x = bix * bdx + tix;
    // int thread_idx_y = biy * bdy + tiy;
    // int thread_x_len = gdx * bdx;
    // int thread_idx = thread_idx_y * thread_x_len + thread_idx_x;
    
    // int num_loops = N / SMEM_PART;

    // A_part[tiy][tix] = A[thread_idx];
    // for(int i = 0; i < num_loops; i++){
    //     C_part[tiy][tix + i * SMEM_PART] = 0.0;
    // }
    // __syncthreads();
    // for(int i = 0; i < num_loops; i++){
    //     B_part[tiy][tix] = B[(i * SMEM_PART + tiy) * K + bix * SMEM_PART + tix];
    //     __syncthreads();
    //     for(int j = 0; j < SMEM_PART; j++){
    //         atomicAdd_double(&C_part[tiy][i * SMEM_PART + (tiy + j) % SMEM_PART], A_part[tiy][tix] * B_part[(tiy + j) % SMEM_PART][tix]);
    //     }
    // }
    
    // for(int i = 0; i < num_loops; i++){
    //     atomicAdd_double(&C[biy * SMEM_PART * N + tiy * N + i * SMEM_PART + tix], C_part[tiy][i * SMEM_PART + tix]);
    // }
}

__global__ void DGEMM_v7(double * A, double * B, double * C){
    __shared__ double A_part[SMEM_PART][SMEM_PART], B_part[SMEM_PART][SMEM_PART];
    __shared__ double C_part[SMEM_PART][SMEM_PART];
    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    int num_loops = K / SMEM_PART;

    C_part[tiy][tix] = 0.0;
    for(int i = 0; i < num_loops; i++){
        int A_glo_x = i * SMEM_PART + tix;
        int A_glo_y = biy * SMEM_PART + tiy;
        int B_glo_x = A_glo_x;
        int B_glo_y = bix * SMEM_PART + tiy;
        A_part[tiy][tix] = A[A_glo_y * K + A_glo_x];
        B_part[tiy][tix] = B[B_glo_y * K + B_glo_x];
        __syncthreads();
        for(int j = 0; j < SMEM_PART; j++){
            atomicAdd_double(&C_part[tiy][(tiy + j) % SMEM_PART], A_part[tiy][tix] * B_part[(tiy + j) % SMEM_PART][tix]);
        }
    }
    int C_glo_x = bix * bdy + tix;
    int C_glo_y = biy * bdx + tiy;
    C[C_glo_y * N + C_glo_x] = C_part[tiy][tix];
}

__inline__ __device__ double warpReduce(double localSum){
    localSum += __shfl_xor(localSum, 16);
    localSum += __shfl_xor(localSum, 8);
    localSum += __shfl_xor(localSum, 4);
    localSum += __shfl_xor(localSum, 2);
    localSum += __shfl_xor(localSum, 1);
    return localSum;
}
// v8是失败的 因为寄存器数量不够
__global__ void DGEMM_v8(double * A, double * B, double * C){
    __shared__ double A_part[SMEM_PART][SMEM_PART], B_part[SMEM_PART][SMEM_PART];
    // __shared__ double C_part[SMEM_PART][SMEM_PART];
    volatile double C_part_reg[SMEM_PART];
    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    int num_loops = K / SMEM_PART;

    // C_part[tiy][tix] = 0.0;
    for(int i = 0; i < num_loops; i++){
        int A_glo_x = i * SMEM_PART + tix;
        int A_glo_y = biy * SMEM_PART + tiy;
        int B_glo_x = A_glo_x;
        int B_glo_y = bix * SMEM_PART + tiy;
        A_part[tiy][tix] = A[A_glo_y * K + A_glo_x];
        B_part[tiy][tix] = B[B_glo_y * K + B_glo_x];
        __syncthreads();
        for(int j = 0; j < SMEM_PART; j++){
            C_part_reg[(tiy + j) % SMEM_PART] += A_part[tiy][tix] * B_part[(tiy + j) % SMEM_PART][tix];
            // atomicAdd_double(&C_part[tiy][(tiy + j) % SMEM_PART], A_part[tiy][tix] * B_part[(tiy + j) % SMEM_PART][tix]);
        }
    }
    for(int i = 0; i < SMEM_PART; i++){
        int C_glo_x = bix * bdy + i;
        int C_glo_y = biy * bdx + tiy;
        atomicAdd_double(&C[C_glo_y * N + C_glo_x], C_part_reg[i]);
        // atomicAdd_double(&C_part[tiy][i], C_part_reg[i]);
        // C_part[tiy][i] += C_part_reg[i];
    }
    
    // // 改reduce
    // for(int i = 0; i < SMEM_PART; i++){
    //     C_part_reg[i] = warpReduce(C_part_reg[i]);
    // }
    // if(tix == 0){
    //     for(int i = 0; i < SMEM_PART; i++){
    //         C_part[tiy][i] = C_part_reg[i];
    //     }
    // }
    
    int C_glo_x = bix * bdy + tix;
    int C_glo_y = biy * bdx + tiy;
    // C[C_glo_y * N + C_glo_x] = C_part[tiy][tix];
}


__host__ void callDGEMM(double * d_A, double * d_B_T, double * d_C, double * d_B){
    int kernelIdx = 7;
    size_t C_Bytes = sizeof(double) * M * N;
    dim3 blockSize(1);
    dim3 gridSize(1);
    switch(kernelIdx){
        case 6:
            blockSize = {SMEM_PART, SMEM_PART};
            gridSize = {K/SMEM_PART, M/SMEM_PART};
            // printf("gridSize:  (%d, %d, %d)\n", gridSize.x, gridSize.y, gridSize.z);
            // printf("blockSize: (%d, %d, %d)\n", blockSize.x, blockSize.y, blockSize.z);
            DGEMM_v6<<<gridSize, blockSize>>>(d_A, d_B_T, d_C);
            break;
        case 7:
            blockSize = {SMEM_PART, SMEM_PART};
            gridSize = {N/SMEM_PART, M/SMEM_PART};
            printf("gridSize:  (%d, %d, %d)\n", gridSize.x, gridSize.y, gridSize.z);
            printf("blockSize: (%d, %d, %d)\n", blockSize.x, blockSize.y, blockSize.z);
            DGEMM_v7<<<gridSize, blockSize>>>(d_A, d_B_T, d_C);
            break;
        case 8:
            blockSize = {SMEM_PART, SMEM_PART};
            gridSize = {N/SMEM_PART, M/SMEM_PART};
            printf("gridSize:  (%d, %d, %d)\n", gridSize.x, gridSize.y, gridSize.z);
            printf("blockSize: (%d, %d, %d)\n", blockSize.x, blockSize.y, blockSize.z);
            DGEMM_v8<<<gridSize, blockSize>>>(d_A, d_B_T, d_C);
            break;
        default:
            break;
    }
    hipDeviceSynchronize();
    printf("DGEMM after: %s\n", hipGetErrorString(hipGetLastError()));
    double * h_C = (double *)malloc(C_Bytes);
    CHECK(hipMemcpy(h_C, d_C, C_Bytes, hipMemcpyDeviceToHost));

    // std::cout << "Matrix C: " << std::endl;
    // for(int i = 0; i < M; i++){
    //     for(int j = 0; j < N; j++){
    //         std::cout << h_C[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double * d_C_cublas, * h_C_cublas;
    CHECK(hipMalloc(&d_C_cublas, C_Bytes));
    h_C_cublas = (double *)malloc(C_Bytes);
    // hipblasStatus_t stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &alpha, d_A, K, d_B, N, &beta, d_C, M);
    hipblasStatus_t stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C_cublas, N);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(h_C_cublas, d_C_cublas, C_Bytes, hipMemcpyDeviceToHost));

    // std::cout << "Matrix C_cublas: " << std::endl;
    // for(int i = 0; i < M; i++){
    //     for(int j = 0; j < N; j++){
    //         std::cout << h_C_cublas[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;

    // checkRes
    int err_cnt = 0;
    bool wrong = false;
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++){
            double err = h_C[i*N+j] - h_C_cublas[i*N+j];
            if(fabs(err) >=  0.001){
                err_cnt++;
                wrong = true;
                // std::cout << "err: [" << i << ", "  << j << "] " << h_C[i*N+j]  << " - " << h_C_cublas[i*N+j] << " = " << err << std::endl;
            }
        }
    }
    if(!wrong){
        std::cout << "gemm success!" << std::endl;
    }
    else{
        std::cout << "gemm error! nums: " << err_cnt << std::endl;
    }
    
    hipblasDestroy(handle);
    CHECK(hipFree(d_C_cublas));
    free(h_C_cublas);
    free(h_C);
}


int main(int argc, char **argv){
    assignData();

    // GPU 计算
    int B_size = K * N;
    double * d_A, * d_B, * d_C, * d_B_T;
    size_t A_Bytes = sizeof(double) * M * K;
    size_t B_Bytes = sizeof(double) * B_size;
    size_t C_Bytes = sizeof(double) * M * N;
    CHECK(hipMalloc(&d_A, A_Bytes));
    CHECK(hipMalloc(&d_B, B_Bytes));
    CHECK(hipMalloc(&d_C, C_Bytes));
    CHECK(hipMalloc(&d_B_T, B_Bytes));
    CHECK(hipMemcpy(d_A, A, A_Bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, B_Bytes, hipMemcpyHostToDevice));
    CHECK(hipMemset(d_C, 0, C_Bytes));
    
    callTranspose(d_B, d_B_T, B_size);
    callDGEMM(d_A, d_B_T, d_C, d_B);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    CHECK(hipFree(d_B_T));

    return 0;
}