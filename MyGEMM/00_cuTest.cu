#include "hip/hip_runtime.h"
#include "../Common.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include <random>
#include <ctime>
#include "hipblas.h"

#define FETCH_FLOAT4(pointer) (reinterpret_cast<float4*>(&(pointer))[0])

// #define VALID

#ifdef VALID

#define M 256
#define N 256
#define K 2048

#else

#define M 4096
#define N 4096
#define K 2048

#endif

#define BLOCK_XY 256

#define SMEM_Y 128
#define SMEM_X 16
#define REG_Y 8
#define REG_X REG_Y

#define NUM_SMEM_DATA_PER_THREAD ((SMEM_Y * SMEM_X) / BLOCK_XY)

float A[M][K], B[K][N], C[M][N];
const float alpha = 1.0;
const float beta = 0.0;

__host__ void assignData(){
    std::default_random_engine e;
    e.seed(time(0));
    std::uniform_real_distribution<float> u(1.5, 4.5);
    // std::cout << "Matrix A: " << std::endl;
    for(size_t i = 0; i < M; i++){
        for(size_t j = 0; j < K; j++){          
    #ifdef VALID
            // A[i][j] = 1; 
            // A[i][j] = (i*2+j)*0.001; 
            A[i][j] = u(e);
    #else
            // A[i][j] = (i*2+j)*0.001;
            A[i][j] = u(e);
    #endif  
            // std::cout << A[i][j] << " ";
        }
        // std::cout << std::endl;
    }
    // std::cout << std::endl; 
    // std::cout << "Matrix B: " << std::endl;
    for(size_t i = 0; i < K; i++){
        for(size_t j = 0; j < N; j++){
    #ifdef VALID
            // B[i][j] = 1; 
            // B[i][j] = (i*2+j)*0.001;
            B[i][j] = u(e);
    #else
            // B[i][j] = (i*2+j)*0.001;
            B[i][j] = u(e);
    #endif  
            // std::cout << B[i][j] << " ";
        }
        // std::cout << std::endl;
    }
    // std::cout << std::endl;  
    memset(C, 0, sizeof(float) * M * N);
}

// 单次寄存器只处理 4*4 的C矩阵  尽可能避免共享内存和寄存器bank冲突
__global__ void SGEMM_v26(float *A, float *B, float *C){
    __shared__ float A_part[2][SMEM_X][SMEM_Y], B_part[2][SMEM_X][SMEM_Y];

    float A_trans_reg[NUM_SMEM_DATA_PER_THREAD];

    float A_part_reg[2][REG_Y];
    float B_part_reg[2][REG_X];
    float C_part_reg[REG_Y][REG_X];
    #pragma unroll
    for(size_t i = 0; i < REG_Y; i++){
        #pragma unroll
        for(size_t j = 0; j < REG_X; j++){
            C_part_reg[i][j] = 0.0;
        }
    }

    size_t tix = threadIdx.x;
    size_t bix = blockIdx.x, biy = blockIdx.y;
    size_t warp_id = tix / 32;
    size_t lane_id = tix % 32;
    // const size_t tile_index_a = (warp_id/4)*32 + ((lane_id%16)/2)*4;
    size_t A_lds_x = ((warp_id >> 2) << 5)  | ((lane_id & 14) << 1);
    // const size_t tile_index_b = (warp_id%4)*16 + (lane_id/16)*8 + (lane_id%2)*4;
    size_t B_lds_x = ((warp_id & 3) << 4) | ((lane_id >> 4) << 3) | ((lane_id & 1) << 2);

    size_t num_loops_shared = K/SMEM_X;
    size_t num_loops_regs = SMEM_X;

    /// 第 0 个SMEM fetch
    size_t A_ldg_y_stride = BLOCK_XY * 4 / SMEM_X;
    #pragma unroll
    for(size_t ldg_cnt = 0; ldg_cnt < SMEM_Y; ldg_cnt += A_ldg_y_stride){
        size_t A_ldg_y = biy * SMEM_Y + ldg_cnt + (tix << 2) / SMEM_X;
        size_t A_ldg_x = (tix << 2) % SMEM_X;
        size_t A_sts_trans_x = ldg_cnt + (tix << 2) / SMEM_X;
        size_t A_sts_trans_y = (tix << 2) % SMEM_X;
        size_t A_trans_reg_idx = ldg_cnt / A_ldg_y_stride * 4;
        FETCH_FLOAT4(A_trans_reg[A_trans_reg_idx]) = FETCH_FLOAT4(A[A_ldg_y * K + A_ldg_x]);
        A_part[0][A_sts_trans_y][A_sts_trans_x] = A_trans_reg[A_trans_reg_idx];
        A_part[0][A_sts_trans_y+1][A_sts_trans_x] = A_trans_reg[A_trans_reg_idx + 1];
        A_part[0][A_sts_trans_y+2][A_sts_trans_x] = A_trans_reg[A_trans_reg_idx + 2];
        A_part[0][A_sts_trans_y+3][A_sts_trans_x] = A_trans_reg[A_trans_reg_idx + 3];
    }
    size_t B_ldg_y_stride = BLOCK_XY * 4 / SMEM_Y;
    #pragma unroll
    for(size_t ldg_cnt = 0; ldg_cnt < SMEM_X; ldg_cnt += B_ldg_y_stride){
        size_t B_ldg_y = ldg_cnt + (tix << 2) / SMEM_Y;
        size_t B_ldg_x = bix * SMEM_Y + (tix << 2) % SMEM_Y;
        size_t B_sts_y = ldg_cnt + (tix << 2) / SMEM_Y;
        size_t B_sts_x = (tix << 2) % SMEM_Y;
        FETCH_FLOAT4(B_part[0][B_sts_y][B_sts_x]) = FETCH_FLOAT4(B[B_ldg_y * N + B_ldg_x]);
    }
    
    __syncthreads();

    for(size_t i = 1; i < num_loops_shared; i++){
        /// 第 i%2 个SMEM读
        #pragma unroll
        for(size_t ldg_cnt = 0; ldg_cnt < SMEM_Y; ldg_cnt += A_ldg_y_stride){
            size_t A_ldg_y = biy * SMEM_Y + ldg_cnt + (tix << 2) / SMEM_X;
            size_t A_ldg_x = i * SMEM_X + (tix << 2) % SMEM_X;
            size_t A_sts_trans_x = ldg_cnt + (tix << 2) / SMEM_X;
            size_t A_sts_trans_y = (tix << 2) % SMEM_X;
            size_t A_trans_reg_idx = ldg_cnt / A_ldg_y_stride * 4;
            FETCH_FLOAT4(A_trans_reg[A_trans_reg_idx]) = FETCH_FLOAT4(A[A_ldg_y * K + A_ldg_x]);
            A_part[i%2][A_sts_trans_y][A_sts_trans_x] = A_trans_reg[A_trans_reg_idx];
            A_part[i%2][A_sts_trans_y+1][A_sts_trans_x] = A_trans_reg[A_trans_reg_idx + 1];
            A_part[i%2][A_sts_trans_y+2][A_sts_trans_x] = A_trans_reg[A_trans_reg_idx + 2];
            A_part[i%2][A_sts_trans_y+3][A_sts_trans_x] = A_trans_reg[A_trans_reg_idx + 3];
        }
        #pragma unroll
        for(size_t ldg_cnt = 0; ldg_cnt < SMEM_X; ldg_cnt += B_ldg_y_stride){
            size_t B_ldg_y = i * SMEM_X + ldg_cnt + (tix << 2) / SMEM_Y;
            size_t B_ldg_x = bix * SMEM_Y + (tix << 2) % SMEM_Y;
            size_t B_sts_y = ldg_cnt + (tix << 2) / SMEM_Y;
            size_t B_sts_x = (tix << 2) % SMEM_Y;
            FETCH_FLOAT4(B_part[i%2][B_sts_y][B_sts_x]) = FETCH_FLOAT4(B[B_ldg_y * N + B_ldg_x]);
        }

        /// 第 (i-1)%2 个SMEM算
        // 第 0 个 regs读
        FETCH_FLOAT4(A_part_reg[0][0]) = FETCH_FLOAT4(A_part[(i-1)%2][0][A_lds_x]);
        FETCH_FLOAT4(A_part_reg[0][4]) = FETCH_FLOAT4(A_part[(i-1)%2][0][A_lds_x+64]);
        FETCH_FLOAT4(B_part_reg[0][0]) = FETCH_FLOAT4(B_part[(i-1)%2][0][B_lds_x]);
        FETCH_FLOAT4(B_part_reg[0][4]) = FETCH_FLOAT4(B_part[(i-1)%2][0][B_lds_x+64]);

        #pragma unroll
        for(size_t k = 1; k < num_loops_regs; k++){
            // 第 k%2 个regs读
            FETCH_FLOAT4(A_part_reg[k%2][0]) = FETCH_FLOAT4(A_part[(i-1)%2][k][A_lds_x]);
            FETCH_FLOAT4(A_part_reg[k%2][4]) = FETCH_FLOAT4(A_part[(i-1)%2][k][A_lds_x+64]);
            FETCH_FLOAT4(B_part_reg[k%2][0]) = FETCH_FLOAT4(B_part[(i-1)%2][k][B_lds_x]);
            FETCH_FLOAT4(B_part_reg[k%2][4]) = FETCH_FLOAT4(B_part[(i-1)%2][k][B_lds_x+64]);

            // 第 (k-1)%2 个regs算
            #pragma unroll
            for(size_t reg_cnt_a = 0; reg_cnt_a < 4; reg_cnt_a++){
                #pragma unroll
                for(size_t reg_cnt_b = 0; reg_cnt_b < 4; reg_cnt_b++){
                    // 待优化
                    C_part_reg[0+reg_cnt_a][0+reg_cnt_b] += A_part_reg[(k-1)%2][reg_cnt_a] * B_part_reg[(k-1)%2][reg_cnt_b];
                    C_part_reg[0+reg_cnt_a][4+reg_cnt_b] += A_part_reg[(k-1)%2][reg_cnt_a] * B_part_reg[(k-1)%2][4+reg_cnt_b];
                    C_part_reg[4+reg_cnt_a][0+reg_cnt_b] += A_part_reg[(k-1)%2][4+reg_cnt_a] * B_part_reg[(k-1)%2][reg_cnt_b];
                    C_part_reg[4+reg_cnt_a][4+reg_cnt_b] += A_part_reg[(k-1)%2][4+reg_cnt_a] * B_part_reg[(k-1)%2][4+reg_cnt_b];
                }
            }
        } 
        // 第 (num_loops_regs-1)%2 个regs算
        #pragma unroll
        for(size_t reg_cnt_a = 0; reg_cnt_a < 4; reg_cnt_a++){
            #pragma unroll
            for(size_t reg_cnt_b = 0; reg_cnt_b < 4; reg_cnt_b++){
                C_part_reg[0+reg_cnt_a][0+reg_cnt_b] += A_part_reg[(num_loops_regs-1)%2][reg_cnt_a] * B_part_reg[(num_loops_regs-1)%2][reg_cnt_b];
                C_part_reg[0+reg_cnt_a][4+reg_cnt_b] += A_part_reg[(num_loops_regs-1)%2][reg_cnt_a] * B_part_reg[(num_loops_regs-1)%2][4+reg_cnt_b];
                C_part_reg[4+reg_cnt_a][0+reg_cnt_b] += A_part_reg[(num_loops_regs-1)%2][4+reg_cnt_a] * B_part_reg[(num_loops_regs-1)%2][reg_cnt_b];
                C_part_reg[4+reg_cnt_a][4+reg_cnt_b] += A_part_reg[(num_loops_regs-1)%2][4+reg_cnt_a] * B_part_reg[(num_loops_regs-1)%2][4+reg_cnt_b];
            }
        }
        __syncthreads();
    }

    /// 第 (num_loops_shared-1)%2 个SMEM算
    // 第 0 个 regs读
    FETCH_FLOAT4(A_part_reg[0][0]) = FETCH_FLOAT4(A_part[(num_loops_shared-1)%2][0][A_lds_x]);
    FETCH_FLOAT4(A_part_reg[0][4]) = FETCH_FLOAT4(A_part[(num_loops_shared-1)%2][0][A_lds_x+64]);
    FETCH_FLOAT4(B_part_reg[0][0]) = FETCH_FLOAT4(B_part[(num_loops_shared-1)%2][0][B_lds_x]);
    FETCH_FLOAT4(B_part_reg[0][4]) = FETCH_FLOAT4(B_part[(num_loops_shared-1)%2][0][B_lds_x+64]);

    #pragma unroll
    for(size_t k = 1; k < num_loops_regs; k++){
        // 第 k%2 个regs读
        FETCH_FLOAT4(A_part_reg[k%2][0]) = FETCH_FLOAT4(A_part[(num_loops_shared-1)%2][k][A_lds_x]);
        FETCH_FLOAT4(A_part_reg[k%2][4]) = FETCH_FLOAT4(A_part[(num_loops_shared-1)%2][k][A_lds_x+64]);
        FETCH_FLOAT4(B_part_reg[k%2][0]) = FETCH_FLOAT4(B_part[(num_loops_shared-1)%2][k][B_lds_x]);
        FETCH_FLOAT4(B_part_reg[k%2][4]) = FETCH_FLOAT4(B_part[(num_loops_shared-1)%2][k][B_lds_x+64]);

        // 第 (k-1)%2 个regs算
        #pragma unroll
        for(size_t reg_cnt_a = 0; reg_cnt_a < 4; reg_cnt_a++){
            #pragma unroll
            for(size_t reg_cnt_b = 0; reg_cnt_b < 4; reg_cnt_b++){
                // 待优化
                C_part_reg[reg_cnt_a][reg_cnt_b] += A_part_reg[(k-1)%2][reg_cnt_a] * B_part_reg[(k-1)%2][reg_cnt_b];
                C_part_reg[reg_cnt_a][4 + reg_cnt_b] += A_part_reg[(k-1)%2][reg_cnt_a] * B_part_reg[(k-1)%2][4 + reg_cnt_b];
                C_part_reg[4 + reg_cnt_a][reg_cnt_b] += A_part_reg[(k-1)%2][4 + reg_cnt_a] * B_part_reg[(k-1)%2][reg_cnt_b];
                C_part_reg[4+reg_cnt_a][4+reg_cnt_b] += A_part_reg[(k-1)%2][4+reg_cnt_a] * B_part_reg[(k-1)%2][4+reg_cnt_b];
            }
        }
    } 
    // 第 (num_loops_regs-1)%2 个regs算
    #pragma unroll
    for(size_t reg_cnt_a = 0; reg_cnt_a < 4; reg_cnt_a++){
        #pragma unroll
        for(size_t reg_cnt_b = 0; reg_cnt_b < 4; reg_cnt_b++){
            C_part_reg[0+reg_cnt_a][0+reg_cnt_b] += A_part_reg[(num_loops_regs-1)%2][reg_cnt_a] * B_part_reg[(num_loops_regs-1)%2][reg_cnt_b];
            C_part_reg[0+reg_cnt_a][4+reg_cnt_b] += A_part_reg[(num_loops_regs-1)%2][reg_cnt_a] * B_part_reg[(num_loops_regs-1)%2][4+reg_cnt_b];
            C_part_reg[4+reg_cnt_a][0+reg_cnt_b] += A_part_reg[(num_loops_regs-1)%2][4+reg_cnt_a] * B_part_reg[(num_loops_regs-1)%2][reg_cnt_b];
            C_part_reg[4+reg_cnt_a][4+reg_cnt_b] += A_part_reg[(num_loops_regs-1)%2][4+reg_cnt_a] * B_part_reg[(num_loops_regs-1)%2][4+reg_cnt_b];
        }
    }

    //// 写回
    #pragma unroll
    for(size_t reg_cnt_a = 0; reg_cnt_a < 4; reg_cnt_a++){
        size_t C_glo_y = biy * SMEM_Y + A_lds_x + reg_cnt_a;
        size_t C_glo_x = bix * SMEM_Y + B_lds_x;
        FETCH_FLOAT4(C[C_glo_y * N + C_glo_x]) = FETCH_FLOAT4(C_part_reg[reg_cnt_a][0]);
        FETCH_FLOAT4(C[C_glo_y * N + C_glo_x + 64]) = FETCH_FLOAT4(C_part_reg[reg_cnt_a][4]);
        FETCH_FLOAT4(C[(C_glo_y + 64) * N + C_glo_x]) = FETCH_FLOAT4(C_part_reg[4+reg_cnt_a][0]);
        FETCH_FLOAT4(C[(C_glo_y + 64) * N + C_glo_x + 64]) = FETCH_FLOAT4(C_part_reg[4+reg_cnt_a][4]);
    }
}

__host__ void callSGEMM(float * d_A, float * d_B, float * d_C){
    size_t C_Bytes = sizeof(float) * M * N;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float * d_C_cublas, * h_C_cublas;
    CHECK(hipMalloc(&d_C_cublas, C_Bytes));
    h_C_cublas = (float *)malloc(C_Bytes);
    // hipblasStatus_t stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &alpha, d_A, K, d_B, N, &beta, d_C, M);
    hipblasStatus_t stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C_cublas, N);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(h_C_cublas, d_C_cublas, C_Bytes, hipMemcpyDeviceToHost));

    // std::cout << "Matrix C_cublas: " << std::endl;
    // for(size_t i = 0; i < M; i++){
    //     for(size_t j = 0; j < N; j++){
    //         std::cout << h_C_cublas[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;


    size_t kernelIdx = 26;
    dim3 blockSize(1);
    dim3 gridSize(1);
    switch(kernelIdx){
        case 26:
            blockSize = {BLOCK_XY};
            gridSize = {N/SMEM_Y, M/SMEM_Y};
            printf("Kernel Num: %lu\n", kernelIdx);
            printf("gridSize:  (%d, %d, %d)\n", gridSize.x, gridSize.y, gridSize.z);
            printf("blockSize: (%d, %d, %d)\n", blockSize.x, blockSize.y, blockSize.z);
            SGEMM_v26<<<gridSize, blockSize>>>(d_A, d_B, d_C);
            break;
        default:
            break;
    }
    hipDeviceSynchronize();
    printf("SGEMM after: %s\n", hipGetErrorString(hipGetLastError()));
    float * h_C = (float *)malloc(C_Bytes);
    CHECK(hipMemcpy(h_C, d_C, C_Bytes, hipMemcpyDeviceToHost));

    // std::cout << "Matrix C: " << std::endl;
    // for(size_t i = 0; i < M; i++){
    //     for(size_t j = 0; j < N; j++){
    //         std::cout << h_C[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;

    // checkRes
    size_t err_cnt = 0;
    bool wrong = false;
    for(size_t i = 0; i < M; i++){
        for(size_t j = 0; j < N; j++){
            float err = h_C[i*N+j] - h_C_cublas[i*N+j];
            if(fabs(err) >=  0.1){
                err_cnt++;
                wrong = true;
                // std::cout << "err: [" << i << ", "  << j << "] " << h_C[i*N+j]  << " - " << h_C_cublas[i*N+j] << " = " << err << std::endl;
    #ifdef VALID
                std::cout << "err: [" << i << ", "  << j << "] " << h_C[i*N+j]  << " - " << h_C_cublas[i*N+j] << " = " << err << std::endl;
    #endif
            }
        }
    }
    if(!wrong){
        std::cout << "gemm success!" << std::endl;
    }
    else{
        std::cout << "gemm error! nums: " << err_cnt << std::endl;
    }
    
    hipblasDestroy(handle);
    CHECK(hipFree(d_C_cublas));
    free(h_C_cublas);
    free(h_C);
}


int main(){
    assignData();

    float *d_A, *d_B, *d_C;
    size_t A_Bytes = sizeof(float) * M * K;
    size_t B_Bytes = sizeof(float) * N * K;
    size_t C_Bytes = sizeof(float) * M * N;
    CHECK(hipMalloc(&d_A, A_Bytes));
    CHECK(hipMalloc(&d_B, B_Bytes));
    CHECK(hipMalloc(&d_C, C_Bytes));
    CHECK(hipMemcpy(d_A, A, A_Bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, B_Bytes, hipMemcpyHostToDevice));
    CHECK(hipMemset(d_C, 0, C_Bytes));

    callSGEMM(d_A, d_B, d_C);

    CHECK(hipFree(d_A)); 
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    return 0;
}



// nvcc -I/opt/kaiProjects/GEMM_kai/Utils -L /usr/local/cuda/lib64 -l cuda -l cublas -o res/00_res 00_cuTest.cu

// nvcc --keep --keep-dir midRes -gencode=arch=compute_86,code=\"sm_86,compute_86\" -I/opt/kaiProjects/GEMM_kai/Utils -L /usr/local/cuda/lib64 -l cuda -l cublas -o res/00_res 00_cuTest.cu
// cuasm --bin2asm midRes/00_cuTest.sm_86.cubin -o midRes/00_cuTest.sm_86.cuasm