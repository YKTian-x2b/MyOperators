#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>
#include <cassert>
#include <cstdio>
#include "hip/hip_runtime_api.h"
#include "hostptr.hpp"
#include "cuptr.hpp"

#define BLK_Y 16
#define BLK_X 32

#define M 1024
#define N 2048

#define FP 4

#define FETCH_FLOAT4(ele) (reinterpret_cast<float4*>(&(ele))[0])

float A[M][N];

void assignData(){
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++){
            A[i][j] = i * N + j;
        }
    }
}

void valid(float *trans_arr_h){
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++){
            assert(abs(A[i][j]-trans_arr_h[j*M+i]) < 0.01);
        }
    }
}

// 待更新
__global__ void Transpose_float4(float *arr, float *trans_arr){
    __shared__ float smem[BLK_Y][BLK_X+1];
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    int ldg_x = bix * bdx + tix;
    int ldg_y = biy * bdy + tiy;
    int idx = tiy * bdx + tix; 
    int lds_y = idx % BLK_Y;
    int lds_x = idx / BLK_Y;
    int stg_x = biy * bdy + lds_y;
    int stg_y = bix * bdx + lds_x;

    smem[tiy][tix] = arr[ldg_y * N + ldg_x];
    __syncthreads();
    trans_arr[stg_y * M + stg_x] = smem[lds_y][lds_x];
}

int main(){
    assignData();
    dim3 BLK_SIZE(BLK_X, BLK_Y);
    dim3 GRD_SIZE(N/BLK_X/FP, M/BLK_Y/FP);
    CuPtr<float> arr_d(M*N);
    checkCudaErrors(hipMemcpy(arr_d.GetPtr(), A, M*N*sizeof(float), hipMemcpyHostToDevice));
    CuPtr<float> trans_arr_d(M*N);
    Transpose<<<GRD_SIZE, BLK_SIZE>>>(arr_d.GetPtr(), trans_arr_d.GetPtr());
    hipDeviceSynchronize();
    HostPtr<float> trans_arr_h;
    trans_arr_d.ToHostPtr(trans_arr_h);
    valid(trans_arr_h.GetPtr());
}

// nvcc -I/opt/kaiProjects/GEMM_kai/Utils -L /usr/local/cuda/lib64 -l cuda -l cublas -o res/trans_4 Transpose_float4.cu