#include "hip/hip_runtime.h"
#include "../Common.cuh"
#include "../Utils/atomicAdd.cu"
#include <iostream>
#include <random>
#include <ctime>
#include "hipblas.h"
#include <string>

// regsPerBlock和regsPerSM都是65536 这些是指32bit的寄存器
// 如果启动 3个block; 一个block 占 256*64个寄存器 存256*32个double
// 如果启动 2个block；一个block 占 256*128个寄存器 存256*64个double
// 如果启动 1个block; 一个block 占 256*256个寄存器 存256*128个double

// 为了用满SM的3*16个warp，限制一个block是32*16个线程 一个SM启动三个block，那么一个block先用1/4regs 256*64个
// 如果是连续两个regs存一个double的话，一个block处理256*32个，就是16个double每线程?

// smemPerBlock是49152=128*128*3 smemPerSM是102400=32*32*100

// transpose操作 要求M/N/K都至少是16 T_BLOCK_X/T_BLOCK_Y
// gemm操作 要求M/N/K要大于等于GEMM_BLOCK_Y

// #define VALID
#ifdef VALID

#define M 32
#define N 32
#define K 384

#else

#define M 512
#define N 512
#define K 3072

#endif

// 转置用的blockDim 
#define T_BLOCK_X 16
#define T_BLOCK_Y 16

// 矩阵乘用的blockDim 
#define GEMM_BLOCK_X 32
#define GEMM_BLOCK_Y 8
// 用来reg_doublebuffering 一次global->shared填充能带来的 shared->reg填充数
#define REG_DB 6

// 共享内存矩阵的X维
#define SMEM_PART_X (GEMM_BLOCK_X * REG_DB)

// 为了避免register bank冲突，我们定义 单个线程处理的元素个数为 4
// REG_Y和REG_X 分别表示 处理SMEM A列和B行时 要启动的线程数
#define REG_PART_Y 4
#define REG_Y (GEMM_BLOCK_Y / REG_PART_Y)
#define REG_PART_X 4
#define REG_X (GEMM_BLOCK_Y / REG_PART_X)
// 那么共需要REGS个warp来处理
#define REGS (REG_Y * REG_X)

const double alpha = 1.0;
const double beta = 0.0;

double A[M][K], B[K][N];
double C[M][N];

__host__ void assignData(){
    std::default_random_engine e;
    e.seed(time(0));
    std::uniform_real_distribution<double> u(1.5, 4.5);
    // std::cout << "Matrix A: " << std::endl;
    for(int i = 0; i < M; i++){
        for(int j = 0; j < K; j++){          
#ifdef VALID
            A[i][j] = 1; 
#else
            // A[i][j] = (i*2+j)*0.001; 
            A[i][j] = u(e);
#endif  
            // std::cout << A[i][j] << " ";
        }
        // std::cout << std::endl;
    }
    // std::cout << std::endl; 
    // std::cout << "Matrix B: " << std::endl;
    for(int i = 0; i < K; i++){
        for(int j = 0; j < N; j++){
#ifdef VALID
            B[i][j] = 1; 
#else
            // A[i][j] = (i*2+j)*0.001; 
            B[i][j] = u(e);
#endif  
            // std::cout << B[i][j] << " ";
        }
        // std::cout << std::endl;
    }
    // std::cout << std::endl;  
    memset(C, 0, sizeof(double) * M * N);
}

__global__ void transpose(double * B, double * B_T){
    __shared__ double smem[T_BLOCK_Y][T_BLOCK_X];
    // 共享内存和线程块的结构是一样的，全局内存和网格的结构使用一样的。

    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    int gdx = gridDim.x, gdy = gridDim.y;

    int thread_idx_x = bix * bdx + tix;
    int thread_idx_y = biy * bdy + tiy;
    int thread_x_len = gdx * bdx;
    int thread_y_len = gdy * bdy;
    // 既是线程索引也是全局内存B的索引
    int thread_idx = thread_idx_y * thread_x_len + thread_idx_x;
    // B_T索引
    int tran_thread_idx = thread_idx_x * thread_y_len + thread_idx_y;

    // 行读列写
    smem[tiy][tix] = B[thread_idx];
    B_T[tran_thread_idx] = smem[tiy][tix];

}

__host__ void checkRes(double * B_T, double * B_T_blas, const int B_size){
    bool wrong = false;
    for(int i = 0; i < B_size; i++){
        double err = B_T[i] - B_T_blas[i];
        if(fabs(err) >=  0.01){
            wrong = true;
            std::cout << "err: " << B_T[i] << " - " << B_T_blas[i] << " = " << fabs(err) << std::endl;
        }
    }
    if(wrong == false){
        std::cout << "transpose success!" << std::endl;
    }
    else{
        std::cout << "transpose error!" << std::endl;
    }
}

__host__ void callTranspose(double * B, double * B_T, const int B_size){
    size_t B_Bytes = sizeof(double) * B_size;

    dim3 blockSize(T_BLOCK_X, T_BLOCK_Y);
    dim3 gridSize(N/T_BLOCK_X, K/T_BLOCK_Y);
    transpose<<<gridSize, blockSize>>>(B, B_T);
    hipDeviceSynchronize();
    double * h_B_T = (double*)malloc(B_Bytes);
    CHECK(hipMemcpy(h_B_T, B_T, B_Bytes, hipMemcpyDeviceToHost));
    // std::cout << "h_B_T: " << std::endl;
    // printMatrix(h_B_T, N, K);
    
    double * B_T_blas;
    CHECK(hipMalloc(&B_T_blas, B_Bytes));
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t stat = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, K, N, &alpha, B, N, &beta, nullptr, N, B_T_blas, K);
    hipDeviceSynchronize();
    double * h_B_T_blas = (double*)malloc(B_Bytes);
    CHECK(hipMemcpy(h_B_T_blas, B_T_blas, B_Bytes, hipMemcpyDeviceToHost));
    // std::cout << "h_B_T_blas: " << std::endl;
    // printMatrix(h_B_T_blas, N,  K);
 
    checkRes(h_B_T, h_B_T_blas, B_size);

    free(h_B_T);
    CHECK(hipFree(B_T_blas));
    hipblasDestroy(handle);
    free(h_B_T_blas);
}

__inline__ __device__ double warpReduce(double localSum){
    localSum += __shfl_xor(localSum, 16);
    localSum += __shfl_xor(localSum, 8);
    localSum += __shfl_xor(localSum, 4);
    localSum += __shfl_xor(localSum, 2);
    localSum += __shfl_xor(localSum, 1);
    return localSum;
}

__inline__ __device__ double warpReduce_half(double localSum){
    localSum += __shfl_xor(localSum, 8);
    localSum += __shfl_xor(localSum, 4);
    localSum += __shfl_xor(localSum, 2);
    localSum += __shfl_xor(localSum, 1);
    return localSum;
}


__inline__ __device__ double warpReduce_quarter(double localSum){
    localSum += __shfl_xor(localSum, 4);
    localSum += __shfl_xor(localSum, 2);
    localSum += __shfl_xor(localSum, 1);
    return localSum;
}

__inline__ __device__ double callWarpReduce(double localSum){
    switch (GEMM_BLOCK_X){
        case 32:    return warpReduce(localSum);
        case 16:    return warpReduce_half(localSum);
        case 8:     return warpReduce_quarter(localSum);
        default:    return warpReduce(localSum);
    } 
}

// v17性能是cublas的不到一半 原因是 计算吞吐和内存吞吐都不足 所以是 latency bound 应该均衡 内存访问和计算间的周期数
__global__ void DGEMM_v17(double * A, double * B, double * C){
    // smem的block上限是128*128*3=3*128*128字节 一个double8个字节 所以一个block的smem最多存 3*16*128个double
    // double buffering + AB数组对半分的情况下 一次处理的A矩阵元素个数为3*4*128个double
    // 用满的情况下，block/SM limit SMEM就是2
    // 用到2*4*128个double，block/SM limit SMEM就是3 受限于SM SMEM总量102400
    __shared__ double A_part[2][GEMM_BLOCK_Y][SMEM_PART_X], B_part[2][GEMM_BLOCK_Y][SMEM_PART_X];

    // 如果启动 2个block；一个block 占 256*128个寄存器 存256*64个double
    // 如果启动 4个block；一个block 占 256*64个寄存器 存256*32个double
    // 2*(REG_PART_Y+REG_PART_X) + REG_PART_Y*REG_PART_X
    volatile double A_part_reg[2][REG_PART_Y];
    volatile double B_part_reg[2][REG_PART_X];
    volatile double C_part_reg[REG_PART_Y][REG_PART_X];

    for(int i = 0; i < REG_PART_Y; i++){
        A_part_reg[0][i] = 0.0;
        A_part_reg[1][i] = 0.0;
        
    }
    for(int i = 0; i < REG_PART_X; i++){
        B_part_reg[0][i] = 0.0;
        B_part_reg[1][i] = 0.0;
    }
    for(int i = 0; i < REG_PART_Y; i++){
        for(int j = 0; j < REG_PART_X; j++){
            C_part_reg[i][j] = 0.0;
        }
    }
    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    // K至少得是 SMEM_PART_X 所以 这里应该有一个if判断 让最后一轮多余的warps continue
    int num_loops = K / SMEM_PART_X;

     //// 读第0个SMEM块
     int A_glo_x = tix;
     int A_glo_y = biy * GEMM_BLOCK_Y + tiy;
     int B_glo_y = bix * GEMM_BLOCK_Y + tiy;
     for(int i = 0; i < REG_DB; i++){
         A_part[0][tiy][tix+i*GEMM_BLOCK_X] = A[A_glo_y * K + A_glo_x + i*GEMM_BLOCK_X];
         B_part[0][tiy][tix+i*GEMM_BLOCK_X] = B[B_glo_y * K + A_glo_x + i*GEMM_BLOCK_X];
     }
     __syncthreads();

    for(int i = 1; i < num_loops; i++){
        //// 读第i%2个SMEM块
        A_glo_x = i * SMEM_PART_X + tix;
        for(int reg_db = 0; reg_db < REG_DB; reg_db++){
            A_part[i%2][tiy][tix+reg_db*GEMM_BLOCK_X] = A[A_glo_y * K + A_glo_x + reg_db*GEMM_BLOCK_X];
            B_part[i%2][tiy][tix+reg_db*GEMM_BLOCK_X] = B[B_glo_y * K + A_glo_x + reg_db*GEMM_BLOCK_X];
        }
        
        //// 算第(i-1)%2个SMEM块

        // 只用tiy==0的那个线程束的前16个线程
        if(tiy == 0){     
        
            // 读第0个A_regs
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                A_part_reg[0][reg_y] = A_part[(i-1)%2][reg_y][tix];
            }
            // 读第0个B_regs
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                B_part_reg[0][reg_x] = B_part[(i-1)%2][reg_x][tix];
            }

            // 读第j%2个A_regs和B_regs 算第j-1个C_regs
            for(int j = 1; j < REG_DB; j++){
                for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                    A_part_reg[j%2][reg_y] = A_part[(i-1)%2][reg_y][tix+j*GEMM_BLOCK_X];
                }
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    B_part_reg[j%2][reg_x] = B_part[(i-1)%2][reg_x][tix+j*GEMM_BLOCK_X];
                }
                for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                    for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                        C_part_reg[reg_y][reg_x] += A_part_reg[(j-1)%2][reg_y] * B_part_reg[(j-1)%2][reg_x];
                    }
                }
            }
            // 算第(REG_DB-1)个C_regs
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    C_part_reg[reg_y][reg_x] += A_part_reg[(REG_DB-1)%2][reg_y] * B_part_reg[(REG_DB-1)%2][reg_x];
                }
            }
        }
        __syncthreads();
    }

    //// 算第(num_loops-1)%2个SMEM块

    if(tiy == 0){
        // 读第0个A_regs
        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            A_part_reg[0][reg_y] = A_part[(num_loops-1)%2][reg_y][tix];
        }
        // 读第0个B_regs
        for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
            B_part_reg[0][reg_x] = B_part[(num_loops-1)%2][reg_x][tix];
        }

        // 读第j%2个A_regs和B_regs 算第j-1个C_regs
        for(int j = 1; j < REG_DB; j++){
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                A_part_reg[j%2][reg_y] = A_part[(num_loops-1)%2][reg_y][tix+j*GEMM_BLOCK_X];
            }
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                B_part_reg[j%2][reg_x] = B_part[(num_loops-1)%2][reg_x][tix+j*GEMM_BLOCK_X];
            }
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    C_part_reg[reg_y][reg_x] += A_part_reg[(j-1)%2][reg_y] * B_part_reg[(j-1)%2][reg_x];
                }
            }
        }
        // 算第(REG_DB-1)个C_regs
        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                C_part_reg[reg_y][reg_x] += A_part_reg[(REG_DB-1)%2][reg_y] * B_part_reg[(REG_DB-1)%2][reg_x];
            }
        }

        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                C_part_reg[reg_y][reg_x] = callWarpReduce(C_part_reg[reg_y][reg_x]);
            }
        }
        
        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            int tmp_tix = tix;
            while(tmp_tix < REG_PART_X){
                int C_glo_y = bdy * biy + reg_y;
                int C_glo_x = bdy * bix + tmp_tix;
                C[C_glo_y * N + C_glo_x] = C_part_reg[reg_y][tmp_tix];
                tmp_tix += GEMM_BLOCK_X;
            }
        }
    }
}

__global__ void DGEMM_v18(double * A, double * B, double * C){
    // smem的block上限是128*128*3=3*128*128字节 一个double8个字节 所以一个block的smem最多存 3*16*128个double
    // double buffering + AB数组对半分的情况下 一次处理的A矩阵元素个数为3*4*128个double
    // 用满的情况下，block/SM limit SMEM就是2
    // 用到2*4*128个double，block/SM limit SMEM就是3 受限于SM SMEM总量102400
    __shared__ double A_part[2][GEMM_BLOCK_Y][SMEM_PART_X], B_part[2][GEMM_BLOCK_Y][SMEM_PART_X];

    // 如果启动 2个block；一个block 占 256*128个寄存器 存256*64个double
    // 如果启动 4个block；一个block 占 256*64个寄存器 存256*32个double
    // 2*(REG_PART_Y+REG_PART_X) + REG_PART_Y*REG_PART_X
    volatile double A_part_reg[2][REG_PART_Y];
    volatile double B_part_reg[2][REG_PART_X];
    volatile double C_part_reg[REG_PART_Y][REG_PART_X];

    for(int i = 0; i < REG_PART_Y; i++){
        A_part_reg[0][i] = 0.0;
        A_part_reg[1][i] = 0.0;
        
    }
    for(int i = 0; i < REG_PART_X; i++){
        B_part_reg[0][i] = 0.0;
        B_part_reg[1][i] = 0.0;
    }
    for(int i = 0; i < REG_PART_Y; i++){
        for(int j = 0; j < REG_PART_X; j++){
            C_part_reg[i][j] = 0.0;
        }
    }
    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    // K至少得是 SMEM_PART_X 所以 这里应该有一个if判断 让最后一轮多余的warps continue
    int num_loops = K / SMEM_PART_X;

     //// 读第0个SMEM块
     int A_glo_x = tix;
     int A_glo_y = biy * GEMM_BLOCK_Y + tiy;
     int B_glo_y = bix * GEMM_BLOCK_Y + tiy;
     for(int i = 0; i < REG_DB; i++){
         A_part[0][tiy][tix+i*GEMM_BLOCK_X] = A[A_glo_y * K + A_glo_x + i*GEMM_BLOCK_X];
         B_part[0][tiy][tix+i*GEMM_BLOCK_X] = B[B_glo_y * K + A_glo_x + i*GEMM_BLOCK_X];
     }
     __syncthreads();

    for(int i = 1; i < num_loops; i++){
        //// 读第i%2个SMEM块
        A_glo_x = i * SMEM_PART_X + tix;
        for(int reg_db = 0; reg_db < REG_DB; reg_db++){
            A_part[i%2][tiy][tix+reg_db*GEMM_BLOCK_X] = A[A_glo_y * K + A_glo_x + reg_db*GEMM_BLOCK_X];
            B_part[i%2][tiy][tix+reg_db*GEMM_BLOCK_X] = B[B_glo_y * K + A_glo_x + reg_db*GEMM_BLOCK_X];
        }
        
        //// 算第(i-1)%2个SMEM块

        if(tiy < REGS){     
        
            // 读第0个A_regs
            // tiy/REG_Y相同的tiy处理相同的A行 tiy%REG_Y相同的tiy处理相同的B列
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                A_part_reg[0][reg_y] = A_part[(i-1)%2][reg_y + (tiy/REG_Y)*REG_PART_Y][tix];
            }
            // 读第0个B_regs
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                B_part_reg[0][reg_x] = B_part[(i-1)%2][reg_x + (tiy%REG_Y)*REG_PART_Y][tix];
            }

            // 读第j%2个A_regs和B_regs 算第j-1个C_regs
            for(int j = 1; j < REG_DB; j++){
                for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                    A_part_reg[j%2][reg_y] = A_part[(i-1)%2][reg_y + (tiy/REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
                }
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    B_part_reg[j%2][reg_x] = B_part[(i-1)%2][reg_x + (tiy%REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
                }
                for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                    for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                        C_part_reg[reg_y][reg_x] += A_part_reg[(j-1)%2][reg_y] * B_part_reg[(j-1)%2][reg_x];
                    }
                }
            }
            // 算第(REG_DB-1)个C_regs
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    C_part_reg[reg_y][reg_x] += A_part_reg[(REG_DB-1)%2][reg_y] * B_part_reg[(REG_DB-1)%2][reg_x];
                }
            }
        }
        __syncthreads();
    }

    //// 算第(num_loops-1)%2个SMEM块

    if(tiy < REGS){
        // 读第0个A_regs
        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            A_part_reg[0][reg_y] = A_part[(num_loops-1)%2][reg_y + (tiy/REG_Y)*REG_PART_Y][tix];
        }
        // 读第0个B_regs
        for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
            B_part_reg[0][reg_x] = B_part[(num_loops-1)%2][reg_x + (tiy%REG_Y)*REG_PART_Y][tix];
        }

        // 读第j%2个A_regs和B_regs 算第j-1个C_regs
        for(int j = 1; j < REG_DB; j++){
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                A_part_reg[j%2][reg_y] = A_part[(num_loops-1)%2][reg_y + (tiy/REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
            }
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                B_part_reg[j%2][reg_x] = B_part[(num_loops-1)%2][reg_x + (tiy%REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
            }
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    C_part_reg[reg_y][reg_x] += A_part_reg[(j-1)%2][reg_y] * B_part_reg[(j-1)%2][reg_x];
                }
            }
        }
        // 算第(REG_DB-1)个C_regs
        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                C_part_reg[reg_y][reg_x] += A_part_reg[(REG_DB-1)%2][reg_y] * B_part_reg[(REG_DB-1)%2][reg_x];
            }
        }

        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                C_part_reg[reg_y][reg_x] = callWarpReduce(C_part_reg[reg_y][reg_x]);
            }
        }
        
        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            int tmp_tix = tix;
            while(tmp_tix < REG_PART_X){
                int C_glo_x = bdy * bix + tmp_tix + (tiy%2)*REG_PART_Y;
                int C_glo_y = bdy * biy + reg_y + (tiy/2)*REG_PART_Y;
                C[C_glo_y * N + C_glo_x] = C_part_reg[reg_y][tmp_tix];
                tmp_tix += GEMM_BLOCK_X;
            }
        }
    }
}

// error
__global__ void DGEMM_v18_NonPrefetch(float *A, float *B, float *C){
    __shared__ float A_part[GEMM_BLOCK_Y][SMEM_PART_X], B_part[GEMM_BLOCK_Y][SMEM_PART_X];
    volatile float A_part_reg[REG_PART_Y];
    volatile float B_part_reg[REG_PART_X];
    volatile float C_part_reg[REG_PART_Y][REG_PART_X];
    for(int i = 0; i < REG_PART_Y; i++){
        A_part_reg[i] = 0.0; 
    }
    for(int i = 0; i < REG_PART_X; i++){
        B_part_reg[i] = 0.0;
    }
    for(int i = 0; i < REG_PART_Y; i++){
        for(int j = 0; j < REG_PART_X; j++){
            C_part_reg[i][j] = 0.0;
        }
    }
    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    int A_glo_x;
     int A_glo_y = biy * GEMM_BLOCK_Y + tiy;
     int B_glo_y = bix * GEMM_BLOCK_Y + tiy;
    // K至少得是 SMEM_PART_X 所以 这里应该有一个if判断 让最后一轮多余的warps continue
    int num_loops = K / SMEM_PART_X;
    for(int i = 0; i < num_loops; i++){
        //// 读SMEM块
        A_glo_x = i * SMEM_PART_X + tix;
        for(int reg_db = 0; reg_db < REG_DB; reg_db++){
            A_part[tiy][tix+reg_db*GEMM_BLOCK_X] = A[A_glo_y * K + A_glo_x + reg_db*GEMM_BLOCK_X];
            B_part[tiy][tix+reg_db*GEMM_BLOCK_X] = B[B_glo_y * K + A_glo_x + reg_db*GEMM_BLOCK_X];
        }
        __syncthreads();
        //// 算SMEM块
        if(tiy < REGS){     
            // 读A_regs和B_regs 算C_regs
            for(int j = 0; j < REG_DB; j++){
                for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                    A_part_reg[reg_y] = A_part[reg_y + (tiy/REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
                }
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    B_part_reg[reg_x] = B_part[reg_x + (tiy%REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
                }
                for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                    for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                        C_part_reg[reg_y][reg_x] += A_part_reg[reg_y] * B_part_reg[reg_x];
                    }
                }
            }
        }
    }
    if(tiy < REGS){
        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                C_part_reg[reg_y][reg_x] = callWarpReduce(C_part_reg[reg_y][reg_x]);
            }
        }
        
        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            int tmp_tix = tix;
            while(tmp_tix < REG_PART_X){
                int C_glo_x = bdy * bix + tmp_tix + (tiy%REG_Y)*REG_PART_Y;
                int C_glo_y = bdy * biy + reg_y + (tiy/REG_Y)*REG_PART_Y;
                C[C_glo_y * N + C_glo_x] = C_part_reg[reg_y][tmp_tix];
                tmp_tix += GEMM_BLOCK_X;
            }
        }
    }
}

// 矩阵C的寄存器计算的二重循环换个顺序 没啥效果
__global__ void DGEMM_v19(float * A, float * B, float * C){
    // smem的block上限是128*128*3=3*128*128字节 一个float4个字节 所以一个block的smem最多存 3*32*128个float
    // fdoublebuffering + AB数组对半分的情况下 一次处理的A矩阵元素个数为3*8*128个float
    // 用满的情况下，block/SM limit SMEM就是2
    __shared__ float A_part[2][GEMM_BLOCK_Y][SMEM_PART_X], B_part[2][GEMM_BLOCK_Y][SMEM_PART_X];

    // 如果启动 2个block；一个block 占 256*128个寄存器 存256*64*2个float
    // 如果启动 4个block；一个block 占 256*64个寄存器 存256*32*2个float
    // 2*(REG_PART_Y+REG_PART_X) + REG_PART_Y*REG_PART_X
    float A_part_reg[2][REG_PART_Y];
    float B_part_reg[2][REG_PART_X];
    float C_part_reg[REG_PART_Y][REG_PART_X];

    for(int i = 0; i < REG_PART_Y; i++){
        A_part_reg[0][i] = 0.0;
        A_part_reg[1][i] = 0.0; 
    }
    for(int i = 0; i < REG_PART_X; i++){
        B_part_reg[0][i] = 0.0;
        B_part_reg[1][i] = 0.0;
    }
    for(int i = 0; i < REG_PART_Y; i++){
        for(int j = 0; j < REG_PART_X; j++){
            C_part_reg[i][j] = 0.0;
        }
    }
    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    // K至少得是 SMEM_PART_X 所以 这里应该有一个if判断 让最后一轮多余的warps continue
    int num_loops = K / SMEM_PART_X;

     //// 读第0个SMEM块
     int A_glo_x = tix;
     int A_glo_y = biy * GEMM_BLOCK_Y + tiy;
     int B_glo_y = bix * GEMM_BLOCK_Y + tiy;
     for(int i = 0; i < REG_DB; i++){
         A_part[0][tiy][tix+i*GEMM_BLOCK_X] = A[A_glo_y * K + A_glo_x + i*GEMM_BLOCK_X];
         B_part[0][tiy][tix+i*GEMM_BLOCK_X] = B[B_glo_y * K + A_glo_x + i*GEMM_BLOCK_X];
     }
     __syncthreads();

    for(int i = 1; i < num_loops; i++){
        //// 读第i%2个SMEM块
        A_glo_x = i * SMEM_PART_X + tix;
        for(int reg_db = 0; reg_db < REG_DB; reg_db++){
            A_part[i%2][tiy][tix+reg_db*GEMM_BLOCK_X] = A[A_glo_y * K + A_glo_x + reg_db*GEMM_BLOCK_X];
            B_part[i%2][tiy][tix+reg_db*GEMM_BLOCK_X] = B[B_glo_y * K + A_glo_x + reg_db*GEMM_BLOCK_X];
        }
        
        //// 算第(i-1)%2个SMEM块
        if(tiy < REGS){     
            // 读第0个A_regs
            // tiy/REG_Y相同的tiy处理相同的A行 tiy%REG_Y相同的tiy处理相同的B列
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                A_part_reg[0][reg_y] = A_part[(i-1)%2][reg_y + (tiy/REG_Y)*REG_PART_Y][tix];
            }
            // 读第0个B_regs
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                B_part_reg[0][reg_x] = B_part[(i-1)%2][reg_x + (tiy%REG_Y)*REG_PART_Y][tix];
            }

            // 读第j%2个A_regs和B_regs 算第j-1个C_regs
            for(int j = 1; j < REG_DB; j++){
                for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                    A_part_reg[j%2][reg_y] = A_part[(i-1)%2][reg_y + (tiy/REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
                }
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    B_part_reg[j%2][reg_x] = B_part[(i-1)%2][reg_x + (tiy%REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
                }
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){  
                        C_part_reg[reg_y][reg_x] += A_part_reg[(j-1)%2][reg_y] * B_part_reg[(j-1)%2][reg_x];
                    }
                }
            }
            // 算第(REG_DB-1)个C_regs
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                    C_part_reg[reg_y][reg_x] += A_part_reg[(REG_DB-1)%2][reg_y] * B_part_reg[(REG_DB-1)%2][reg_x];
                }
            }
        }
        __syncthreads();
    }

    //// 算第(num_loops-1)%2个SMEM块
    if(tiy < REGS){
        // 读第0个A_regs
        for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
            A_part_reg[0][reg_y] = A_part[(num_loops-1)%2][reg_y + (tiy/REG_Y)*REG_PART_Y][tix];
        }
        // 读第0个B_regs
        for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
            B_part_reg[0][reg_x] = B_part[(num_loops-1)%2][reg_x + (tiy%REG_Y)*REG_PART_Y][tix];
        }

        // 读第j%2个A_regs和B_regs 算第j-1个C_regs
        for(int j = 1; j < REG_DB; j++){
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                A_part_reg[j%2][reg_y] = A_part[(num_loops-1)%2][reg_y + (tiy/REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
            }
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                B_part_reg[j%2][reg_x] = B_part[(num_loops-1)%2][reg_x + (tiy%REG_Y)*REG_PART_Y][tix+j*GEMM_BLOCK_X];
            }
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                    C_part_reg[reg_y][reg_x] += A_part_reg[(j-1)%2][reg_y] * B_part_reg[(j-1)%2][reg_x];
                }
            }
        }
        // 算第(REG_DB-1)个C_regs
        for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                C_part_reg[reg_y][reg_x] += A_part_reg[(REG_DB-1)%2][reg_y] * B_part_reg[(REG_DB-1)%2][reg_x];
            }
        }
        for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){    
                C_part_reg[reg_y][reg_x] = callWarpReduce(C_part_reg[reg_y][reg_x]);
            }
        }
        
        // 把结果写回全局内存
        if(tix < REG_PART_Y*REG_PART_X){
            int C_reg_x = tix%REG_PART_Y;
            int C_reg_y = tix/REG_PART_Y;
            int C_glo_x = bdy*bix + (tiy%REG_Y)*REG_PART_Y + C_reg_x;
            int C_glo_y = bdy*biy + (tiy/REG_Y)*REG_PART_Y + C_reg_y;
            C[C_glo_y * N + C_glo_x] = C_part_reg[C_reg_y][C_reg_x];
        }
    }
}

__host__ void callDGEMM(double * d_A, double * d_B_T, double * d_C, double * d_B){
    int kernelIdx = 18;
    size_t C_Bytes = sizeof(double) * M * N;
    dim3 blockSize(1);
    dim3 gridSize(1);
    switch(kernelIdx){
        case 17:
            blockSize = {GEMM_BLOCK_X, GEMM_BLOCK_Y};
            gridSize = {N/GEMM_BLOCK_Y, M/GEMM_BLOCK_Y};
            printf("Kernel Num: %d\n", kernelIdx);
            printf("gridSize:  (%d, %d, %d)\n", gridSize.x, gridSize.y, gridSize.z);
            printf("blockSize: (%d, %d, %d)\n", blockSize.x, blockSize.y, blockSize.z);
            DGEMM_v17<<<gridSize, blockSize>>>(d_A, d_B_T, d_C);
            break;
        case 18:
            blockSize = {GEMM_BLOCK_X, GEMM_BLOCK_Y};
            gridSize = {N/GEMM_BLOCK_Y, M/GEMM_BLOCK_Y};
            printf("Kernel Num: %d\n", kernelIdx);
            printf("gridSize:  (%d, %d, %d)\n", gridSize.x, gridSize.y, gridSize.z);
            printf("blockSize: (%d, %d, %d)\n", blockSize.x, blockSize.y, blockSize.z);
            DGEMM_v18<<<gridSize, blockSize>>>(d_A, d_B_T, d_C);
            break;
        default:
            break;
    }
    hipDeviceSynchronize();
    printf("DGEMM after: %s\n", hipGetErrorString(hipGetLastError()));
    double * h_C = (double *)malloc(C_Bytes);
    CHECK(hipMemcpy(h_C, d_C, C_Bytes, hipMemcpyDeviceToHost));

    // std::cout << "Matrix C: " << std::endl;
    // for(int i = 0; i < M; i++){
    //     for(int j = 0; j < N; j++){
    //         std::cout << h_C[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double * d_C_cublas, * h_C_cublas;
    CHECK(hipMalloc(&d_C_cublas, C_Bytes));
    h_C_cublas = (double *)malloc(C_Bytes);
    // hipblasStatus_t stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &alpha, d_A, K, d_B, N, &beta, d_C, M);
    hipblasStatus_t stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C_cublas, N);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(h_C_cublas, d_C_cublas, C_Bytes, hipMemcpyDeviceToHost));

    // std::cout << "Matrix C_cublas: " << std::endl;
    // for(int i = 0; i < M; i++){
    //     for(int j = 0; j < N; j++){
    //         std::cout << h_C_cublas[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;

    // checkRes
    int err_cnt = 0;
    bool wrong = false;
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++){
            double err = h_C[i*N+j] - h_C_cublas[i*N+j];
            if(fabs(err) >=  0.001){
                err_cnt++;
                wrong = true;
#ifdef VALID
                std::cout << "err: [" << i << ", "  << j << "] " << h_C[i*N+j]  << " - " << h_C_cublas[i*N+j] << " = " << err << std::endl;
#endif
            }
        }
    }
    if(!wrong){
        std::cout << "gemm success!" << std::endl;
    }
    else{
        std::cout << "gemm error! nums: " << err_cnt << std::endl;
    }
    
    hipblasDestroy(handle);
    CHECK(hipFree(d_C_cublas));
    free(h_C_cublas);
    free(h_C);
}


int main(int argc, char **argv){
    assignData();

    // GPU 计算
    int B_size = K * N;
    double * d_A, * d_B, * d_C, * d_B_T;
    size_t A_Bytes = sizeof(double) * M * K;
    size_t B_Bytes = sizeof(double) * B_size;
    size_t C_Bytes = sizeof(double) * M * N;
    CHECK(hipMalloc(&d_A, A_Bytes));
    CHECK(hipMalloc(&d_B, B_Bytes));
    CHECK(hipMalloc(&d_C, C_Bytes));
    CHECK(hipMalloc(&d_B_T, B_Bytes));
    CHECK(hipMemcpy(d_A, A, A_Bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, B_Bytes, hipMemcpyHostToDevice));
    CHECK(hipMemset(d_C, 0, C_Bytes));
    
    callTranspose(d_B, d_B_T, B_size);
    callDGEMM(d_A, d_B_T, d_C, d_B);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    CHECK(hipFree(d_B_T));

    return 0;
}