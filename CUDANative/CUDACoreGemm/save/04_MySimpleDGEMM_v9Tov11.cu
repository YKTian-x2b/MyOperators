#include "hip/hip_runtime.h"
#include "../Common.cuh"
#include "../Utils/atomicAdd.cu"
#include <iostream>
#include <random>
#include <ctime>
#include "hipblas.h"
#include <string>

// transpose操作 要求M/N/K都至少是16

#ifdef VALID

#define M 32
#define N 32
#define K 128

#else

#define M 512
#define N 512
#define K 2048
// #define M 128
// #define N 128
// #define K 4096

#endif

// M/N/K要大于等于SMEM_PART
#define SMEM_PART_X 32
#define SMEM_PART_Y 16

#define REG_PART_Y 2
#define REG_PART_X 2

#define T_BLOCK_X 16
#define T_BLOCK_Y 16

const double alpha = 1.0;
const double beta = 0.0;

double A[M][K], B[K][N];
double C[M][N];

__host__ void assignData(){
    std::default_random_engine e;
    e.seed(time(0));
    std::uniform_real_distribution<double> u(1.5, 4.5);
    // std::cout << "Matrix A: " << std::endl;
    for(int i = 0; i < M; i++){
        for(int j = 0; j < K; j++){
            // A[i][j] = u(e);
            // A[i][j] = (i*2+j)*0.001; 
            A[i][j] = 1;   
            // std::cout << A[i][j] << " ";
        }
        // std::cout << std::endl;
    }
    // std::cout << std::endl; 
    // std::cout << "Matrix B: " << std::endl;
    for(int i = 0; i < K; i++){
        for(int j = 0; j < N; j++){
            // B[i][j] = u(e);
            // B[i][j] = (i*2+j)*0.001;   
            B[i][j] = 1; 
            // std::cout << B[i][j] << " ";
        }
        // std::cout << std::endl;
    }
    // std::cout << std::endl;  
    memset(C, 0, sizeof(double) * M * N);
}

__global__ void transpose(double * B, double * B_T){
    __shared__ double smem[T_BLOCK_Y][T_BLOCK_X];
    // 共享内存和线程块的结构是一样的，全局内存和网格的结构使用一样的。

    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    int gdx = gridDim.x, gdy = gridDim.y;

    int thread_idx_x = bix * bdx + tix;
    int thread_idx_y = biy * bdy + tiy;
    int thread_x_len = gdx * bdx;
    int thread_y_len = gdy * bdy;
    // 既是线程索引也是全局内存B的索引
    int thread_idx = thread_idx_y * thread_x_len + thread_idx_x;
    // B_T索引
    int tran_thread_idx = thread_idx_x * thread_y_len + thread_idx_y;

    // 行读列写
    smem[tiy][tix] = B[thread_idx];
    B_T[tran_thread_idx] = smem[tiy][tix];

}

__host__ void checkRes(double * B_T, double * B_T_blas, const int B_size){
    bool wrong = false;
    for(int i = 0; i < B_size; i++){
        double err = B_T[i] - B_T_blas[i];
        if(fabs(err) >=  0.01){
            wrong = true;
            std::cout << "err: " << B_T[i] << " - " << B_T_blas[i] << " = " << fabs(err) << std::endl;
        }
    }
    if(wrong == false){
        std::cout << "transpose success!" << std::endl;
    }
    else{
        std::cout << "transpose error!" << std::endl;
    }
}

__host__ void callTranspose(double * B, double * B_T, const int B_size){
    size_t B_Bytes = sizeof(double) * B_size;

    dim3 blockSize(T_BLOCK_X, T_BLOCK_Y);
    dim3 gridSize(N/T_BLOCK_X, K/T_BLOCK_Y);
    transpose<<<gridSize, blockSize>>>(B, B_T);
    hipDeviceSynchronize();
    double * h_B_T = (double*)malloc(B_Bytes);
    CHECK(hipMemcpy(h_B_T, B_T, B_Bytes, hipMemcpyDeviceToHost));
    // std::cout << "h_B_T: " << std::endl;
    // printMatrix(h_B_T, N, K);
    
    double * B_T_blas;
    CHECK(hipMalloc(&B_T_blas, B_Bytes));
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasStatus_t stat = hipblasDgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, K, N, &alpha, B, N, &beta, nullptr, N, B_T_blas, K);
    hipDeviceSynchronize();
    double * h_B_T_blas = (double*)malloc(B_Bytes);
    CHECK(hipMemcpy(h_B_T_blas, B_T_blas, B_Bytes, hipMemcpyDeviceToHost));
    // std::cout << "h_B_T_blas: " << std::endl;
    // printMatrix(h_B_T_blas, N,  K);
 
    checkRes(h_B_T, h_B_T_blas, B_size);

    free(h_B_T);
    CHECK(hipFree(B_T_blas));
    hipblasDestroy(handle);
    free(h_B_T_blas);
}

__inline__ __device__ double warpReduce(double localSum){
    localSum += __shfl_xor(localSum, 16);
    localSum += __shfl_xor(localSum, 8);
    localSum += __shfl_xor(localSum, 4);
    localSum += __shfl_xor(localSum, 2);
    localSum += __shfl_xor(localSum, 1);
    return localSum;
}

// 1.8 times longer than cublas
__global__ void DGEMM_v9(double * A, double * B, double * C){
    // 每个block负责 A矩阵某SMEM_PART_Y行和B矩阵某SMEM_PART_Y列构成的 两个子矩阵
    __shared__ double A_part[SMEM_PART_Y][SMEM_PART_X], B_part[SMEM_PART_Y][SMEM_PART_X];
    // 每个线程的寄存器存储 block负责的子矩阵 smemA的tix行 和 smemB所有列 的乘积元素
    volatile double C_part_reg[SMEM_PART_Y];
    for(int i = 0; i < SMEM_PART_Y; i++){
        C_part_reg[i] = 0.0;
    }
    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    // 再将子矩阵沿 K 划分为多个循环，每个循环处理一个 SMEM_PART_Y*SMEM_PART_X
    int num_loops = K / SMEM_PART_X;
    #pragma unroll
    for(int i = 0; i < num_loops; i++){
        int A_glo_x = i * SMEM_PART_X + tix;
        int A_glo_y = biy * SMEM_PART_Y + tiy;
        int B_glo_y = bix * SMEM_PART_Y + tiy;
        A_part[tiy][tix] = A[A_glo_y * K + A_glo_x];
        B_part[tiy][tix] = B[B_glo_y * K + A_glo_x];
        __syncthreads();
        // num_loops的每次迭代都会将子矩阵 smemA的tix行 和 smemB所有列 的一个块 累加到 寄存器。
        for(int j = 0; j < SMEM_PART_Y; j++){
            C_part_reg[(tiy + j) % SMEM_PART_Y] += A_part[tiy][tix] * B_part[(tiy + j) % SMEM_PART_Y][tix];
        }
    }
    // 然后进行所有寄存器结果的warpreduce 结果是同一warp每个线程的所有寄存器都存有相同的结果元素
    for(int i = 0; i < SMEM_PART_Y; i++){
        C_part_reg[i] = warpReduce(C_part_reg[i]);
    }
    // 仅将 SMEM_PART_Y * SMEM_PART_Y 的结果矩阵写回
    if(tix < SMEM_PART_Y){
        int C_glo_y = biy * bdy + tiy;
        int C_glo_x = bix * bdy + tix;
        C[C_glo_y * N + C_glo_x] = C_part_reg[tix];
    }
}

// smem double buffering  1.9 times longer than cublas
// perf mode is error
__global__ void DGEMM_v10(double * A, double * B, double * C){
    __shared__ double A_part[2][SMEM_PART_Y][SMEM_PART_X], B_part[2][SMEM_PART_Y][SMEM_PART_X];
    volatile double C_part_reg[SMEM_PART_Y];
    for(int i = 0; i < SMEM_PART_Y; i++){
        C_part_reg[i] = 0.0;
    }
    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    int num_loops = K / SMEM_PART_X;
    // 读取A_part B_part
    int A_glo_x = tix;
    int A_glo_y = biy * SMEM_PART_Y + tiy;
    int B_glo_y = bix * SMEM_PART_Y + tiy;
    A_part[0][tiy][tix] = A[A_glo_y * K + A_glo_x];
    B_part[0][tiy][tix] = B[B_glo_y * K + A_glo_x];
    // 确保读到了
    __syncthreads();
    // 
    #pragma unroll
    for(int i = 1; i < num_loops; i++){
        // 读下一项
        A_glo_x = i * SMEM_PART_X + tix;
        A_part[i%2][tiy][tix] = A[A_glo_y * K + A_glo_x];
        B_part[i%2][tiy][tix] = B[B_glo_y * K + A_glo_x];
        // 计算当前项
        for(int j = 0; j < SMEM_PART_Y; j++){
            C_part_reg[(tiy + j) % SMEM_PART_Y] += A_part[(i-1)%2][tiy][tix] * B_part[(i-1)%2][(tiy + j) % SMEM_PART_Y][tix];
        }
        __syncthreads();
    }
    for(int j = 0; j < SMEM_PART_Y; j++){
        C_part_reg[(tiy + j) % SMEM_PART_Y] += A_part[1][tiy][tix] * B_part[1][(tiy + j) % SMEM_PART_Y][tix];
    }
    // 
    for(int i = 0; i < SMEM_PART_Y; i++){
        C_part_reg[i] = warpReduce(C_part_reg[i]);
    }
    if(tix < SMEM_PART_Y){
        int C_glo_y = biy * bdy + tiy;
        int C_glo_x = bix * bdy + tix;
        C[C_glo_y * N + C_glo_x] = C_part_reg[tix];
    }
}

// #define SMEM_PART_X 32
// #define SMEM_PART_Y 16
// #define REG_PART_Y 2
// #define REG_PART_X 2
// register分块
__global__ void DGEMM_v11(double * A, double * B, double * C){
    __shared__ double A_part[SMEM_PART_Y][SMEM_PART_X], B_part[SMEM_PART_Y][SMEM_PART_X];
    volatile double A_part_reg[REG_PART_Y];
    volatile double B_part_reg[REG_PART_X];
    volatile double C_part_reg[REG_PART_Y][REG_PART_X];
    volatile double C_loss_reg[REG_PART_Y][REG_PART_X];
    for(int i = 0; i < SMEM_PART_Y; i++){
        A_part_reg[i] = 0.0;
    }
    for(int i = 0; i < SMEM_PART_X; i++){
        B_part_reg[i] = 0.0;
    }
    for(int i = 0; i < SMEM_PART_Y; i++){
        for(int j = 0; j < SMEM_PART_X; j++){
            C_part_reg[i][j] = 0.0;
            C_loss_reg[i][j] = 0.0;
        }
    }

    // 线程组索引
    int tix = threadIdx.x, tiy = threadIdx.y;
    int bix = blockIdx.x, biy = blockIdx.y;
    int bdx = blockDim.x, bdy = blockDim.y;
    int num_loops = K / SMEM_PART_X;

    for(int i = 0; i < num_loops; i++){
        int A_glo_x = i * SMEM_PART_X + tix;
        int A_glo_y = biy * SMEM_PART_Y + tiy;
        int B_glo_y = bix * SMEM_PART_Y + tiy;
        A_part[tiy][tix] = A[A_glo_y * K + A_glo_x];
        B_part[tiy][tix] = B[B_glo_y * K + A_glo_x];
        __syncthreads();
        if(tiy*REG_PART_Y >= SMEM_PART_Y || tix*REG_PART_X >= SMEM_PART_X)
            continue;
        
        for(int j = 0; j < SMEM_PART_X; j++){
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                int A_smem_y = reg_y + tiy * REG_PART_Y;
                A_part_reg[reg_y] = A_part[A_smem_y][j];
            }
            for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                int B_smem_y = reg_x + tix * REG_PART_X;
                B_part_reg[reg_x] = B_part[B_smem_y][j];
            }
            for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
                for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
                    // double y = A_part_reg[reg_y] * B_part_reg[reg_x] - C_loss_reg[reg_y][reg_x];
                    // double t = C_part_reg[reg_y][reg_x] + y;
                    // C_loss_reg[reg_y][reg_x] = (t-C_part_reg[reg_y][reg_x])-y;
                    // C_part_reg[reg_y][reg_x] = t;
                    
                    // // 两个double乘的计算错误导致报错: 非法内存访问
                    // C_part_reg[reg_y][reg_x] = A_part_reg[reg_y] * B_part_reg[reg_x];
                }
            }
        }
    }
    if(tiy*REG_PART_Y >= SMEM_PART_Y || tix*REG_PART_X >= SMEM_PART_X)
        return;
    for(int reg_y = 0; reg_y < REG_PART_Y; reg_y++){
        for(int reg_x = 0; reg_x < REG_PART_X; reg_x++){
            int C_glo_y = biy * bdy + tiy * REG_PART_Y + reg_y;
            int C_glo_x = bix * bdx + tix * REG_PART_X + reg_x;
            C[C_glo_y * N + C_glo_x] = C_part_reg[reg_y][reg_x];
        }
    }
}

__host__ void callDGEMM(double * d_A, double * d_B_T, double * d_C, double * d_B){
    int kernelIdx = 11;
    size_t C_Bytes = sizeof(double) * M * N;
    dim3 blockSize(1);
    dim3 gridSize(1);
    switch(kernelIdx){
        case 9:
            blockSize = {SMEM_PART_X, SMEM_PART_Y};
            gridSize = {N/SMEM_PART_Y, M/SMEM_PART_Y};
            printf("Kernel Num: %d\n", kernelIdx);
            printf("gridSize:  (%d, %d, %d)\n", gridSize.x, gridSize.y, gridSize.z);
            printf("blockSize: (%d, %d, %d)\n", blockSize.x, blockSize.y, blockSize.z);
            DGEMM_v9<<<gridSize, blockSize>>>(d_A, d_B_T, d_C);
            break;
        case 10:
            blockSize = {SMEM_PART_X, SMEM_PART_Y};
            gridSize = {N/SMEM_PART_Y, M/SMEM_PART_Y};
            printf("Kernel Num: %d\n", kernelIdx);
            printf("gridSize:  (%d, %d, %d)\n", gridSize.x, gridSize.y, gridSize.z);
            printf("blockSize: (%d, %d, %d)\n", blockSize.x, blockSize.y, blockSize.z);
            DGEMM_v10<<<gridSize, blockSize>>>(d_A, d_B_T, d_C);
            break;
        case 11:
            blockSize = {SMEM_PART_X, SMEM_PART_Y};
            gridSize = {N/SMEM_PART_Y, M/SMEM_PART_Y};
            printf("Kernel Num: %d\n", kernelIdx);
            printf("gridSize:  (%d, %d, %d)\n", gridSize.x, gridSize.y, gridSize.z);
            printf("blockSize: (%d, %d, %d)\n", blockSize.x, blockSize.y, blockSize.z);
            DGEMM_v11<<<gridSize, blockSize>>>(d_A, d_B_T, d_C);
            break;
        default:
            break;
    }
    hipDeviceSynchronize();
    printf("DGEMM after: %s\n", hipGetErrorString(hipGetLastError()));
    double * h_C = (double *)malloc(C_Bytes);
    CHECK(hipMemcpy(h_C, d_C, C_Bytes, hipMemcpyDeviceToHost));

    // std::cout << "Matrix C: " << std::endl;
    // for(int i = 0; i < M; i++){
    //     for(int j = 0; j < N; j++){
    //         std::cout << h_C[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    double * d_C_cublas, * h_C_cublas;
    CHECK(hipMalloc(&d_C_cublas, C_Bytes));
    h_C_cublas = (double *)malloc(C_Bytes);
    // hipblasStatus_t stat = hipblasDgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, N, K, &alpha, d_A, K, d_B, N, &beta, d_C, M);
    hipblasStatus_t stat = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, d_B, N, d_A, K, &beta, d_C_cublas, N);
    hipDeviceSynchronize();
    CHECK(hipMemcpy(h_C_cublas, d_C_cublas, C_Bytes, hipMemcpyDeviceToHost));

    // std::cout << "Matrix C_cublas: " << std::endl;
    // for(int i = 0; i < M; i++){
    //     for(int j = 0; j < N; j++){
    //         std::cout << h_C_cublas[i*N+j] << " ";
    //     }
    //     std::cout << std::endl;
    // }
    // std::cout << std::endl;

    // checkRes
    int err_cnt = 0;
    bool wrong = false;
    for(int i = 0; i < M; i++){
        for(int j = 0; j < N; j++){
            double err = h_C[i*N+j] - h_C_cublas[i*N+j];
            if(fabs(err) >=  0.001){
                err_cnt++;
                wrong = true;
                std::cout << "err: [" << i << ", "  << j << "] " << h_C[i*N+j]  << " - " << h_C_cublas[i*N+j] << " = " << err << std::endl;
            }
        }
    }
    if(!wrong){
        std::cout << "gemm success!" << std::endl;
    }
    else{
        std::cout << "gemm error! nums: " << err_cnt << std::endl;
    }
    
    hipblasDestroy(handle);
    CHECK(hipFree(d_C_cublas));
    free(h_C_cublas);
    free(h_C);
}


int main(int argc, char **argv){
    assignData();

    // GPU 计算
    int B_size = K * N;
    double * d_A, * d_B, * d_C, * d_B_T;
    size_t A_Bytes = sizeof(double) * M * K;
    size_t B_Bytes = sizeof(double) * B_size;
    size_t C_Bytes = sizeof(double) * M * N;
    CHECK(hipMalloc(&d_A, A_Bytes));
    CHECK(hipMalloc(&d_B, B_Bytes));
    CHECK(hipMalloc(&d_C, C_Bytes));
    CHECK(hipMalloc(&d_B_T, B_Bytes));
    CHECK(hipMemcpy(d_A, A, A_Bytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B, B, B_Bytes, hipMemcpyHostToDevice));
    CHECK(hipMemset(d_C, 0, C_Bytes));
    
    callTranspose(d_B, d_B_T, B_size);
    callDGEMM(d_A, d_B_T, d_C, d_B);

    CHECK(hipFree(d_A));
    CHECK(hipFree(d_B));
    CHECK(hipFree(d_C));
    CHECK(hipFree(d_B_T));

    return 0;
}