#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdio>
#include <cstring>
#include <cstdlib>
#include <random>
#include <ctime>
#include "../Common.cuh"

#define BDIMX 32

__inline__ __device__ int warpReduce(int localSum)
{
    localSum += __shfl_xor(localSum, 16);
    localSum += __shfl_xor(localSum, 8);
    localSum += __shfl_xor(localSum, 4);
    localSum += __shfl_xor(localSum, 2);
    localSum += __shfl_xor(localSum, 1);
    return localSum;
}

__global__ void testShuffle(int * input, int * output){
    size_t bid = blockIdx.x;
    size_t tid = threadIdx.x;
    size_t idx = tid;
    int mySum = input[idx];
    mySum = warpReduce(mySum);
    output[idx] = mySum;
}

int main() {
    int size_bytes = sizeof(int) * BDIMX;
    int * h_in = (int*)malloc(size_bytes);
    int * h_out = (int*)malloc(size_bytes);
    for(int i = 0; i < BDIMX; i++){
        h_in[i] = i + 1;
    }
    int * d_in, *d_out;
    hipMalloc(&d_in, size_bytes);
    hipMalloc(&d_out, size_bytes);

    hipMemcpy(d_in, h_in, size_bytes, hipMemcpyHostToDevice);

    testShuffle<<<1, BDIMX>>>(d_in, d_out);

    hipMemcpy(h_out, d_out, size_bytes, hipMemcpyDeviceToHost);
    for(int i = 0; i < BDIMX; i++){
        printf("%d ", h_out[i]);
    }
    printf("\n");
    hipFree(d_in);
    hipFree(d_out);
    free(h_in);
    free(h_out);
    return 0;
}