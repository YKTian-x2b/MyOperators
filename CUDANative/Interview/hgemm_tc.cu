#include "hip/hip_runtime.h"
template<> class fragment<matrix_a, 16, 16, 16, __half, row_major> : public __frag_base<__half, 16> {};
template<> class fragment<matrix_a, 16, 16, 16, __half, col_major> : public __frag_base<__half, 16> {};
template<> class fragment<matrix_b, 16, 16, 16, __half, row_major> : public __frag_base<__half, 16> {};
template<> class fragment<matrix_b, 16, 16, 16, __half, col_major> : public __frag_base<__half, 16> {};
template<> class fragment<accumulator, 16, 16, 16, __half> : public __frag_base<__half, 8> {};
template<> class fragment<accumulator, 16, 16, 16, float> : public __frag_base<float, 8> {};


__CUDA_MMA_DEVICE_DECL__ void load_matrix_sync(fragment<matrix_a, 16, 16, 16, __half, row_major>& a, const __half* p, unsigned ldm) __DEF_IF_HOST
__CUDA_MMA_DEVICE_DECL__ void load_matrix_sync(fragment<matrix_a, 16, 16, 16, __half, col_major>& a, const __half* p, unsigned ldm) __DEF_IF_HOST
__CUDA_MMA_DEVICE_DECL__ void load_matrix_sync(fragment<matrix_b, 16, 16, 16, __half, row_major>& a, const __half* p, unsigned ldm) __DEF_IF_HOST
__CUDA_MMA_DEVICE_DECL__ void load_matrix_sync(fragment<matrix_b, 16, 16, 16, __half, col_major>& a, const __half* p, unsigned ldm) __DEF_IF_HOST
__CUDA_MMA_DEVICE_DECL__ void load_matrix_sync(fragment<accumulator, 16, 16, 16, __half>& a, const __half* p, unsigned ldm, layout_t layout) __DEF_IF_HOST
__CUDA_MMA_DEVICE_DECL__ void load_matrix_sync(fragment<accumulator, 16, 16, 16, float>& a, const float* p, unsigned ldm, layout_t layout) __DEF_IF_HOST


__CUDA_MMA_DEVICE_DECL__ void store_matrix_sync(__half *p, const fragment<accumulator, 16, 16, 16, __half>& a, unsigned ldm, layout_t layout) __DEF_IF_HOST
__CUDA_MMA_DEVICE_DECL__ void store_matrix_sync(float *p, const fragment<accumulator, 16, 16, 16, float>& a, unsigned ldm, layout_t layout) __DEF_IF_HOST
                 

__CUDA_MMA_DEVICE_DECL__ void mma_sync(fragment<accumulator,16, 16, 16, __half>& d, const fragment<matrix_a, 16, 16, 16, __half, row_major>& a, const fragment<matrix_b,16, 16, 16, __half, col_major>& b, const fragment<accumulator,16, 16, 16, float>& c) __DEF_IF_HOST
__CUDA_MMA_DEVICE_DECL__ void mma_sync(fragment<accumulator,16, 16, 16, __half>& d, const fragment<matrix_a, 16, 16, 16, __half, col_major>& a, const fragment<matrix_b,16, 16, 16, __half, col_major>& b, const fragment<accumulator,16, 16, 16, float>& c) __DEF_IF_HOST
__CUDA_MMA_DEVICE_DECL__ void mma_sync(fragment<accumulator,16, 16, 16, __half>& d, const fragment<matrix_a, 16, 16, 16, __half, row_major>& a, const fragment<matrix_b,16, 16, 16, __half, row_major>& b, const fragment<accumulator,16, 16, 16, float>& c) __DEF_IF_HOST
__CUDA_MMA_DEVICE_DECL__ void mma_sync(fragment<accumulator,16, 16, 16, __half>& d, const fragment<matrix_a, 16, 16, 16, __half, col_major>& a, const fragment<matrix_b,16, 16, 16, __half, row_major>& b, const fragment<accumulator,16, 16, 16, float>& c) __DEF_IF_HOST


void fill_fragment(__frag_base<FragEleType, size, packed_size> &f, 
    const typename helper_traits<FragEleType>::fill_argument_type &in);